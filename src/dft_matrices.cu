#include "hip/hip_runtime.h"
/*
 * Copyright (c) Microsoft Corporation.
 * Licensed under the MIT License.
 */
#include <stdio.h>
#include "numeric_types.h"
#include "basis_func.h"
#include "molecular_integral.h"
#include "cuda_helper.h"

extern "C"
{
    __global__ void cg_electron_repulsion_integral_cuda_kernel(struct basis_func* basis_func_buff, u32* basis_func_index_buff, u64 count, f64* result)
    {
        u64 i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < count)
        {
            result[i] = cg_electron_repulsion_integral(basis_func_buff + basis_func_index_buff[i * 4],
                        basis_func_buff + basis_func_index_buff[i * 4 + 1],
                        basis_func_buff + basis_func_index_buff[i * 4 + 2],
                        basis_func_buff + basis_func_index_buff[i * 4 + 3]);
        }
    }

    void cg_electron_repulsion_integral_cuda(struct basis_func* basis_func_buff, u32* basis_func_index_buff, u64 count, f64* result)
    {
        u64 device_count = (u64)cuda_get_device_count();

        u64 task_count[MAX_GPU_COUNT];
        for (u64 i = 0; i < device_count; ++i)
        {
            task_count[i] = count / device_count;
        }
        for (u64 i = 0; i < count % device_count; ++i)
        {
            task_count[i] += 1;
        }

        u64 offset = 0;
        for (u64 i = 0; i < device_count; ++i)
        {
            hipSetDevice(i);
            cg_electron_repulsion_integral_cuda_kernel<<<(task_count[i] + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK, THREAD_PER_BLOCK>>>(basis_func_buff, &basis_func_index_buff[4 * offset], task_count[i], &result[offset]);
            offset += task_count[i];
        }

        cuda_sync_all();
    }
}
